#include "hip/hip_runtime.h"
#include <iostream>
#include <set>
#include "headers/read_csv.h"
#include "headers/read_fasta.h"
#include <thread>
#include <vector>
#include <chrono>
#include <atomic>
#include <string.h>
#include <hip/hip_runtime_api.h>

inline std::chrono::high_resolution_clock::time_point get_current_time_fenced() {
    std::atomic_thread_fence(std::memory_order_seq_cst);
    auto res_time = std::chrono::high_resolution_clock::now();
    std::atomic_thread_fence(std::memory_order_seq_cst);
    return res_time;
}

template<class D>
inline long long to_us(const D &d) {
    return std::chrono::duration_cast<std::chrono::microseconds>(d).count();
}


void readAllFasta(concurrent_que<std::string> *q) {
    auto start = get_current_time_fenced();
    std::string path = "../data/archive/pseudo88.fasta";
//    for (const auto &entry: fs::directory_iterator(path)) {
    read_fasta(path, q);
//    }
    q->push(" ");
    auto end = get_current_time_fenced();
}

__global__
void matchWords(const char *str, size_t *matched, aho_corasick::Node *dVec, int size) {

    aho_corasick::Node *node;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int iter = i; iter < size; iter += stride) {
        node = &dVec[0];
        while (i < size) {
            char letter = str[i];
            if (letter == 'N') {
                break;
            }

            int nodesIndex = node->children[abs((letter - 65) / 2 % 5 - 1)];
            if (nodesIndex == 0) {
                break;
            }

            node = &dVec[nodesIndex];

            if (node->markerHash) {
                matched[node->markerId] = node->markerHash;
            }
            ++i;
        }

        i = iter + stride;
    }
}


int main() {

    concurrent_que<std::string> qFasta{};

    // read csv
    auto start_t = get_current_time_fenced();

    std::vector<std::string> st;
    auto markers_start = get_current_time_fenced();
    read_csv("../data/markers.csv", st);
    auto markers_end = get_current_time_fenced();
    std::cout << "Finished reading csv\n";


    // read fasta
    auto readFastaStart = get_current_time_fenced();

    std::thread fastaThread{readAllFasta, &qFasta};
    fastaThread.join();

    auto readFastaEnd = get_current_time_fenced();

    std::cout << "Fasta read time : " << to_us(readFastaEnd - readFastaStart) << std::endl;

    auto c1 = get_current_time_fenced();
    auto *a = new aho_corasick::AhoCorasick(st);
    auto c2 = get_current_time_fenced();

    std::cout << "Build trie time : " << to_us(c2 - c1) << std::endl;

    std::cout << "Size of nodes in bytes: " << a->nodeNum * sizeof(aho_corasick::Node) << "\n";

    std::string fasta = qFasta.pop();

    std::cout << fasta.size() << " fasta size\n";
    char *cudaStr;
    size_t *cudaMatched;
    aho_corasick::Node *devNodes;

    auto startCopying = get_current_time_fenced();
    if (hipMalloc((void **) &devNodes, a->nodeNum * sizeof(aho_corasick::Node)) == hipErrorOutOfMemory) {
        exit(1);
    }
    hipMemcpy(devNodes, a->nodes, a->nodeNum * sizeof(aho_corasick::Node), hipMemcpyHostToDevice);


    int blockSize = 256;
    int numBlocks = ((int) fasta.size() + blockSize - 1) / blockSize;


    auto startMatching = get_current_time_fenced();

    hipMallocManaged((void **) &cudaStr, fasta.size());
    hipMallocManaged((void **) &cudaMatched, st.size() * sizeof(size_t));
    hipMemcpy(cudaStr, fasta.data(), fasta.size(), hipMemcpyHostToDevice);
    hipMemset(cudaMatched, 0, st.size() * sizeof(size_t));

    matchWords << < numBlocks, blockSize >> > (cudaStr, cudaMatched, devNodes, fasta.size());

    hipDeviceSynchronize();


    auto *matches = (size_t *) malloc(st.size() * sizeof(size_t));
    if (matches == nullptr) {
        std::cout << "Allocation failed\n";
    }
    hipMemcpy(matches, cudaMatched, st.size() * sizeof(size_t), hipMemcpyDeviceToHost);

    auto endMatching = get_current_time_fenced();

    int count = 0;
    for (int i = 0; i < st.size(); ++i) {
        if (matches[i] != 0) {
            count += a->markerIdMap[matches[i]].size();
        }
    }
    auto end_t = get_current_time_fenced();

    std::cout << "count : " << count << "\n";

    std::cout << to_us(startMatching - startCopying) << " time to copy\n";


    std::cout << "Markers: " << st.size() << "\n";
    std::cout << "Trie build time: " << (double) to_us(c2 - c1) / 1000000.0 << "\n";
    std::cout << "Mathcing time: " << (double) to_us(endMatching - startMatching) / 1000000.0 << "\n";
    std::cout << "Total time: " << (double) to_us(end_t - start_t) / 1000000.0 << "\n";
    std::cout << "Read markers.csv time: " << (double) to_us(markers_end - markers_start) / 1000000.0 << "\n";


    delete (a);
    delete (matches);
    hipFree(devNodes);
    hipFree(cudaStr);
    hipFree(cudaMatched);
    hipProfilerStop();
    return 0;
}

