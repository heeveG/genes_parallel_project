#include "hip/hip_runtime.h"
#include <iostream>
#include <set>
#include "headers/read_csv.h"
#include "headers/read_fasta.h"
#include <thread>
#include <vector>
#include <chrono>
#include <atomic>
#include <string.h>
#include <hip/hip_runtime_api.h>

inline std::chrono::high_resolution_clock::time_point get_current_time_fenced() {
    std::atomic_thread_fence(std::memory_order_seq_cst);
    auto res_time = std::chrono::high_resolution_clock::now();
    std::atomic_thread_fence(std::memory_order_seq_cst);
    return res_time;
}

template<class D>
inline long long to_us(const D &d) {
    return std::chrono::duration_cast<std::chrono::microseconds>(d).count();
}


void readAllFasta(concurrent_que<std::string> *q) {
    auto start = get_current_time_fenced();
    std::string path = "data/archive/pseudo88.fasta";
//    for (const auto &entry: fs::directory_iterator(path)) {
    read_fasta(path, q);
//    }
    q->push(" ");
    auto end = get_current_time_fenced();
}

__global__
void matchWords(const char *str, int *matched, aho_corasick::Node *dVec[], int size) {
    aho_corasick::Node *node = dVec[0];
//    printf("TEST: %d\n", dVec[5]->retsCount);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int index;
    for (int iter = i; iter < size; iter += stride) {
        node = dVec[0];
        while (i < size) {
            char letter = str[i];
            switch (letter) {

                case 'A':
                    index = 0;
                    break;
                case 'C':
                    index = 1;
                    break;
                case 'G':
                    index = 2;
                    break;
                case 'T':
                    index = 3;
                    break;
                default:
                    index = -1;

            }
            if (index == -1)
                break;
            int nodesIndex = node->cudaChildren[index];
            if (nodesIndex == 0) {
                break;
            }
            node = dVec[nodesIndex];

            for (int j = 0; j < node->retsCount; ++j) {
                matched[node->cudaRets[j]] = 1;
            }
            ++i;
        }
        i = iter + stride;
    }
}

int main() {

    concurrent_que<std::string> qFasta{};

    // read csv

    std::vector<std::string> st;
    read_csv("data/markers.csv", st);

    std::cout << "Finished reading csv\n";


    // read fasta

    std::thread fastaThread{readAllFasta, &qFasta};
    fastaThread.join();


    auto *a = new aho_corasick::AhoCorasick(st);
    std::cout << "Built trie\n";



    std::string fasta = qFasta.pop();
    char *cudaStr;
    int *cudaMatched;

    auto startCopying = get_current_time_fenced();

    hipMallocManaged(&cudaStr, fasta.size());
    hipMallocManaged(&cudaMatched, st.size() * sizeof(int));


    hipMemcpy(cudaStr, fasta.data(), fasta.size(), hipMemcpyHostToDevice);
    hipMemset(cudaMatched, 0, st.size() * sizeof(int));

    auto **nodesPtr = (aho_corasick::Node **) malloc(a->nodes.size() * sizeof(aho_corasick::Node *));

    int i = 0;

    for (auto &node: a->nodes) {

        auto *newNode = new aho_corasick::Node(' ');

        hipMalloc((void **) &(newNode->cudaChildren), sizeof(int) * node->children.size());
        hipMemcpy(newNode->cudaChildren, node->children.data(), sizeof(int) * node->children.size(),
                   hipMemcpyHostToDevice);
        hipMalloc((void **) &(newNode->cudaRets), sizeof(int) * node->retVals.size());
        hipMemcpy(newNode->cudaRets, node->retVals.data(), sizeof(int) * node->retVals.size(),
                   hipMemcpyHostToDevice);

        newNode->retsCount = node->retVals.size();

        aho_corasick::Node *devNode;

        hipMalloc((void **) &devNode, sizeof(aho_corasick::Node));
        hipMemcpy(devNode, newNode, sizeof(aho_corasick::Node), hipMemcpyHostToDevice);

        nodesPtr[i++] = devNode;
    }


    aho_corasick::Node **devNodes;

    hipMalloc((void ***) &devNodes, a->nodes.size() * sizeof(aho_corasick::Node *));
    hipMemcpy(devNodes, nodesPtr, a->nodes.size() * sizeof(aho_corasick::Node *), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = ((int) fasta.size() + blockSize - 1) / blockSize;


    std::cout << "Copied \n";

    auto startMatching = get_current_time_fenced();

    matchWords << < numBlocks, blockSize >> > (cudaStr, cudaMatched, devNodes, fasta.size());

    hipDeviceSynchronize();

    auto endMatching = get_current_time_fenced();


    int matches[st.size()];
    hipMemcpy(&matches[0], cudaMatched, st.size() * sizeof(int), hipMemcpyDeviceToHost);



//    for (int i = 0; i < 6000; ++i) {
//        if (matches[i] == 1) {
//            std::cout << matches[i] << " at " << i << "\n";
//        }
//
//    }

    std::cout << to_us(startMatching - startCopying) << " time to copy\n";

    std::cout << to_us(endMatching - startMatching) << " time to match\n";


    delete (a);
    delete (nodesPtr);
    hipFree(devNodes);
    hipFree(cudaStr);
    hipFree(cudaMatched);
    hipProfilerStop();
    return 0;
}

