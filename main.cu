#include "hip/hip_runtime.h"
#include <iostream>
#include <set>
#include "headers/read_csv.h"
#include "headers/read_fasta.h"
#include <thread>
#include <vector>
#include <chrono>
#include <atomic>
#include <string.h>
#include <hip/hip_runtime_api.h>

inline std::chrono::high_resolution_clock::time_point get_current_time_fenced() {
    std::atomic_thread_fence(std::memory_order_seq_cst);
    auto res_time = std::chrono::high_resolution_clock::now();
    std::atomic_thread_fence(std::memory_order_seq_cst);
    return res_time;
}

template<class D>
inline long long to_us(const D &d) {
    return std::chrono::duration_cast<std::chrono::microseconds>(d).count();
}


void readAllFasta(concurrent_que<std::string> *q) {
    auto start = get_current_time_fenced();
    std::string path = "../data/archive/pseudo88.fasta";
//    for (const auto &entry: fs::directory_iterator(path)) {
    read_fasta(path, q);
//    }
    q->push(" ");
    auto end = get_current_time_fenced();
}

__global__
void matchWords(const char *str, size_t *matched, aho_corasick::Node *dVec, int size) {

    aho_corasick::Node *node;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int index;
    for (int iter = i; iter < size; iter += stride) {
        node = &dVec[0];
        while (i < size) {
            char letter = str[i];
            index = (letter - 65) / 2;
            if (index == 6) {
                break;
            }

            int nodesIndex = node->children[abs(index % 5 - 1)];
            if (nodesIndex == 0) {
                break;
            }

            node = &dVec[nodesIndex];

            if (node->markerHash) {
                matched[node->markerId] = node->markerHash;
            }
            ++i;
        }

        i = iter + stride;
    }
}


int main() {

    concurrent_que<std::string> qFasta{};

    // read csv

    std::vector<std::string> st;
    read_csv("../data/markers.csv", st);

    std::cout << "Finished reading csv\n";


    // read fasta
    auto readFastaStart = get_current_time_fenced();

    std::thread fastaThread{readAllFasta, &qFasta};
    fastaThread.join();

    auto readFastaEnd = get_current_time_fenced();

    std::cout << "Fasta read time : " << to_us(readFastaEnd - readFastaStart) << std::endl;

    auto c1 = get_current_time_fenced();
    auto *a = new aho_corasick::AhoCorasick(st);
    auto c2 = get_current_time_fenced();

    std::cout << "Build trie time : " << to_us(c2 - c1) << std::endl;

    std::cout << "Size of nodes in bytes: " << a->nodeNum * sizeof(aho_corasick::Node) << "\n";

    std::string fasta = qFasta.pop();

    std::cout << fasta.size() << " fasta size\n";
    char *cudaStr;
    size_t *cudaMatched;
    aho_corasick::Node *devNodes;
    auto startCopying = get_current_time_fenced();

    hipMallocManaged((void **) &cudaStr, fasta.size());
    hipMallocManaged((void **) &cudaMatched, st.size() * sizeof(size_t));


    hipMemcpy(cudaStr, fasta.data(), fasta.size(), hipMemcpyHostToDevice);
    hipMemset(cudaMatched, 0, st.size() * sizeof(size_t));

    if (hipMalloc((void **) &devNodes, a->nodeNum * sizeof(aho_corasick::Node)) == hipErrorOutOfMemory) {
        exit(1);
    }
    hipMemcpy(devNodes, a->nodes, a->nodeNum * sizeof(aho_corasick::Node), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = ((int) fasta.size() + blockSize - 1) / blockSize;

    std::cout << "Copied \n";

    auto startMatching = get_current_time_fenced();

    matchWords << < numBlocks, blockSize >> > (cudaStr, cudaMatched, devNodes, fasta.size());

    hipDeviceSynchronize();

    auto endMatching = get_current_time_fenced();


    auto *matches = (size_t *) malloc(st.size() * sizeof(size_t));
    if (matches == nullptr) {
        std::cout << "Allocation failed\n";
    }
    hipMemcpy(matches, cudaMatched, st.size() * sizeof(size_t), hipMemcpyDeviceToHost);


    int count = 0;
    for (int i = 0; i < st.size(); ++i) {
        if (matches[i] != 0) {
            count += a->markerIdMap[matches[i]].size();
        }
    }

    std::cout << "count : " << count << "\n";

    std::cout << to_us(startMatching - startCopying) << " time to copy\n";

    std::cout << to_us(endMatching - startMatching) << " time to match\n";


    delete (a);
    delete (matches);
    hipFree(devNodes);
    hipFree(cudaStr);
    hipFree(cudaMatched);
    hipProfilerStop();
    return 0;
}

