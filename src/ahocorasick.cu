#include "hip/hip_runtime.h"
//
// Created by heeve on 27.03.20.
//

#include <unordered_map>
#include <set>
#include <queue>
#include <string>
#include <thrust/device_vector.h>


namespace aho_corasick {


    class Node {

        typedef Node *node_ptr;

    public:
        char value;
        int *cudaRets;
        int *cudaChildren;

        int retsCount;
        std::vector<int> retVals;
        std::vector<int> children;


        Node(char init_val)
                : value(init_val), retVals(), retsCount(0) {
            children.resize(4);
            std::fill(children.begin(), children.end(), 0);
        };

//        ~Node() {
//            if (!children.empty()) {
//                for (int i = 0; i < children.size(); ++i) {
//                    if (children[i] != nullptr)
//                        delete (children[i]);
//                }
//            }
//        }


        void addReturnValue(int added) {
            retVals.push_back(added);
        }

        int childIndex(char letter) {
            switch (letter) {
                case 'A':
                    return 0;
                case 'C':
                    return 1;
                case 'G':
                    return 2;
                case 'T':
                    return 3;
                default:
                    return -1;
            }
        }

        node_ptr addChild(const char letter, std::vector<node_ptr> &nodes) {
            auto childIndx = childAt(letter);
            if (childIndx == 0) {
                int index = childIndex(letter);

                assert(index != -1); // childIndex method should always handle all possible inputs

                int newItem = nodes.size();
                node_ptr newNode = new Node(letter);
                nodes.push_back(newNode);
                children[index] = newItem;
                return newNode;
            }
            return nodes[childIndx];
        }


        int childAt(const char &letter) {
            int index = childIndex(letter);
            int childIndx = children[index];
            if (index != -1 && childIndx != 0) {
                return childIndx;
            }
            return 0;
        }


    };

    class AhoCorasick {
        typedef Node *node_ptr;
    public:

        node_ptr start = new Node('&');


        std::vector<std::string> markers;


        std::vector<node_ptr> nodes{1};
        unsigned long longestMarker = 0;

        AhoCorasick() {
        }

        explicit AhoCorasick(std::vector<std::string> &markersInit) :
                markers(markersInit) {

            setUpTrie();
        }

        ~AhoCorasick() {
            delete (start);
        }


        void setUpTrie() {
            nodes.reserve(2 * markers.size());
            nodes[0] = start;
            // Add nodes to the trie
            node_ptr node;
            for (auto &markerItem: markers) {

                int markerId;
                try {
                    markerId = std::stoi(markerItem.substr(0, markerItem.find(',')));
                }
                catch (std::invalid_argument) {
                    continue;
                }
                std::string marker = markerItem.substr(markerItem.find(',') + 1);
                longestMarker = std::max(longestMarker, marker.size());
                if (marker.empty())
                    continue;
                node = start;
                for (const auto &ch: marker) {
                    node = node->addChild(ch, nodes);
                }
                node->addReturnValue(markerId);

            }
        }
    };
};

