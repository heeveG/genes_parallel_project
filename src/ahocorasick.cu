#include "hip/hip_runtime.h"
//
// Created by heeve on 27.03.20.
//

#include <unordered_map>
#include <set>
#include <queue>
#include <string>
#include <thrust/device_vector.h>

namespace aho_corasick {
    class Node {

    public:
        int children[4];
        size_t markerHash = 0;
        int markerId;

        Node() {
            memset(children, 0, 4 * sizeof(int));
        };


        void addReturnValue(size_t hash, int id) {
            markerHash = hash;
            markerId = id;
        }

        int childIndex(char letter) {
            return abs(letter % 65 / 2 % 5 - 1);
        }

        int addChild(const char letter, int &size) {
            auto childIndx = childAt(letter);
            if (childIndx == 0) {
                int index = childIndex(letter);
                children[index] = size;
                return size;
            }
            return childIndx;
        }


        int childAt(const char &letter) {
            int index = childIndex(letter);
            int childIndx = children[index];
            if (index != -1 && childIndx != 0) {
                return childIndx;
            }
            return 0;
        }


    };

    class AhoCorasick {

    public:
        std::vector<std::string> markers;
        int nodeNum = 0;
        int capacity = 1;
        Node *nodes = new Node[capacity];

        std::unordered_map<size_t, std::set<int>> markerIdMap;

        AhoCorasick() {
        }

        explicit AhoCorasick(std::vector<std::string> &markersInit) :
                markers(markersInit) {

            setUpTrie();

        }

        ~AhoCorasick() {
            delete[] (nodes);
        }


        void setUpTrie() {
            reserve(2 * markers.size());
            addNode();
            std::hash<std::string> hash_fn;
//            // Add nodes to the trie
            for (auto &markerItem: markers) {
                if (markerItem.empty())
                    continue;
                int markerId;

                markerId = std::stoi(markerItem.substr(0, markerItem.find(',')));

                std::string marker = markerItem.substr(markerItem.find(',') + 1);

                if (marker.empty())
                    continue;

                int next = 0;

                size_t hash = hash_fn(marker);
                if (markerIdMap.find(hash) == markerIdMap.end()) {
                    for (const auto &ch: marker) {
                        next = nodes[next].addChild(ch, nodeNum);
                        if (next == nodeNum) {
                            addNode();
                        }
                    }
                    nodes[next].addReturnValue(hash, markerId);
                }

                markerIdMap[hash].insert(markerId);
            }
        };

        void reserve(int num) {
            if (capacity < num) {
                Node *newNodes = new Node[num + 1];
                std::copy(nodes, nodes + nodeNum + 1, newNodes);
                delete[]nodes;
                nodes = newNodes;
                capacity = num + 1;
            }

        }

        void addNode() {
            Node *newNode = new Node();
            if (nodeNum == capacity) {
                reserve(2 * capacity);
            }

            nodes[nodeNum] = *newNode;
            ++nodeNum;
        }
    };

}
